#include "hip/hip_runtime.h"
#include <stdint.h>
#include <stdlib.h>
#include <string.h>

#include "opt_2dhisto.h"
#include "ref_2dhisto.h"
#include "util.h"
#include <algorithm>
#include <cutil.h>
#include <math.h>

#define PAD_WIDTH ((INPUT_WIDTH + 128) & 0xFFFFFF80)
#define INPUT_SIZE (INPUT_WIDTH*INPUT_HEIGHT)
#define TILE_SIZE (32)


__global__ void Baseline(uint32_t* dInput, uint32_t* dBins)
{
    int col = blockDim.x * blockIdx.x + threadIdx.x;
    int row = blockDim.y * blockIdx.y + threadIdx.y;

    if (row < INPUT_HEIGHT && col < INPUT_WIDTH) {
        uint32_t index = dInput[col + (row * PAD_WIDTH)];
        atomicAdd(&dBins[index], 1);
    }
}

__global__ void ParallelStrategyOne(uint32_t* dInput, uint32_t* dBins, size_t rectangle_width, size_t rectangle_height) {
    int col = blockIdx.x * rectangle_width;
    int row = blockIdx.y * rectangle_height; 
   
    uint32_t privateHist[HISTO_HEIGHT * HISTO_WIDTH] = {0};
    for (size_t i = 0; i < rectangle_height; i++){
        for (size_t j = 0; j < rectangle_width; j++){
            size_t segmentCol = col + j;
            size_t segmentRow = row + i; 
            if (segmentCol < INPUT_WIDTH && segmentRow < INPUT_HEIGHT){
                uint32_t index = dInput[segmentCol + segmentRow * PAD_WIDTH];
                if (index < HISTO_HEIGHT * HISTO_WIDTH){
                    privateHist[index]++;
                }              
            }           
        }
    }

    for (uint32_t idx = 0; idx < HISTO_HEIGHT * HISTO_WIDTH; idx++) {
        atomicAdd(&(dBins[idx]), privateHist[idx]); // Update global histogram safely
    }
}

__global__ void ParallelStrategyTwo(uint32_t* dInput, uint32_t* dBins)
{
    __shared__ uint32_t shared_hist[HISTO_HEIGHT * HISTO_WIDTH];

    int col = blockDim.x * blockIdx.x + threadIdx.x;
    int row = blockDim.y * blockIdx.y + threadIdx.y;

    // initialize the shared memory bins
    for (int i = threadIdx.x; i < HISTO_HEIGHT * HISTO_WIDTH;
         i = i + blockDim.x) {
        shared_hist[threadIdx.x] = 0;
    }
    __syncthreads();  // initialization synced

    if (row < INPUT_HEIGHT && col < INPUT_WIDTH) {
        uint32_t index = dInput[col + (row * PAD_WIDTH)];
        atomicAdd(&dBins[index], 1);
    }
    __syncthreads();  // final synced (shared memory is now updated)

    // copy results back to global memory from shared memory
    for (int i = threadIdx.x; i < HISTO_HEIGHT * HISTO_WIDTH;
         i = i + blockDim.x) {
        atomicAdd(&dBins[i], shared_hist[i]);
    }
}

__global__ void ParallelStrategyThree(uint32_t* dInput, uint32_t* dBins)
{
    __shared__ uint32_t shared_hist[255];  
    __shared__ uint32_t input_tile[32 * 32];
    
    int col = blockDim.x * blockIdx.x + threadIdx.x;
    int row = blockDim.y * blockIdx.y + threadIdx.y;
    int tid = threadIdx.y * blockDim.x + threadIdx.x;
    
    // Initialize shared memory
    for (int i = tid; i < 255; i += blockDim.x * blockDim.y) {
        shared_hist[i] = 0;
    }
    __syncthreads();
    
    // Load input tile
    if (row < INPUT_HEIGHT && col < INPUT_WIDTH) {
        input_tile[tid] = dInput[col + (row * PAD_WIDTH)];
    }
    __syncthreads();
    
    // Process pixel
    if (row < INPUT_HEIGHT && col < INPUT_WIDTH) {
        uint32_t value = input_tile[tid];
        atomicAdd(&dBins[value], 1);  
    }
}

__global__ void ParallelStrategyFour(uint32_t* dInput, uint32_t* dBins)
{
    __shared__ uint32_t shared_hist[255];
    __shared__ uint32_t input_tile[33 * 32];  // Added padding to avoid bank conflicts
    
    int col = blockDim.x * blockIdx.x + threadIdx.x;
    int row = blockDim.y * blockIdx.y + threadIdx.y;
    int tid = threadIdx.y * blockDim.x + threadIdx.x;
    
    // Initialize shared memory
    for (int i = tid; i < 255; i += blockDim.x * blockDim.y) {
        shared_hist[i] = 0;
    }
    __syncthreads();
    
    // Load input tile with using padded indexing
    if (row < INPUT_HEIGHT && col < INPUT_WIDTH) {
        input_tile[tid + (threadIdx.y)] = dInput[col + (row * PAD_WIDTH)];  // Add threadIdx.y for padding
    }
    __syncthreads();
    
    // Process pixel 
    if (row < INPUT_HEIGHT && col < INPUT_WIDTH) {
        uint32_t value = input_tile[tid + (threadIdx.y)];  // Access with padding offset
        atomicAdd(&dBins[value], 1);
    }
}

__global__ void ParallelStrategyFive(uint32_t* dInput, uint32_t* dBins)
{
    __shared__ uint32_t shared_hist[255];
    __shared__ uint32_t input_tile[33 * 32];
    
    // Calculate global linear index for coalesced access
    int blockOffset = blockIdx.x * blockDim.x + blockIdx.y * blockDim.y * PAD_WIDTH;
    int threadOffset = threadIdx.x + threadIdx.y * PAD_WIDTH;
    int globalIdx = blockOffset + threadOffset;
    
    int col = blockDim.x * blockIdx.x + threadIdx.x;
    int row = blockDim.y * blockIdx.y + threadIdx.y;
    int tid = threadIdx.y * blockDim.x + threadIdx.x;
    
    // Initialize shared memory
    for (int i = tid; i < 255; i += blockDim.x * blockDim.y) {
        shared_hist[i] = 0;
    }
    __syncthreads();
    
    // Load input tile using coalesced access pattern
    if (row < INPUT_HEIGHT && col < INPUT_WIDTH) {
        input_tile[tid + (threadIdx.y)] = dInput[globalIdx];
    }
    __syncthreads();
    
    // Process pixel
    if (row < INPUT_HEIGHT && col < INPUT_WIDTH) {
        uint32_t value = input_tile[tid + (threadIdx.y)];
        atomicAdd(&dBins[value], 1);
    }
}

__global__ void ParallelStrategySix(uint32_t* dInput, uint32_t* dBins) 
{
    __shared__ uint32_t input_tile[49 * 48];  // TILE == 48 (TILE 64 FAILED), 48 is a good limit 
    
    // Calculate global linear index for coalesced access
    int blockOffset = blockIdx.x * blockDim.x + blockIdx.y * blockDim.y * PAD_WIDTH;
    int threadOffset = threadIdx.x + threadIdx.y * PAD_WIDTH;
    int globalIdx = blockOffset + threadOffset;
    
    int col = blockDim.x * blockIdx.x + threadIdx.x;
    int row = blockDim.y * blockIdx.y + threadIdx.y;
    int tid = threadIdx.y * blockDim.x + threadIdx.x;
    
    // Load input tile using coalesced access pattern
    if (row < INPUT_HEIGHT && col < INPUT_WIDTH) {
        input_tile[tid + (threadIdx.y)] = dInput[globalIdx];
    }
    __syncthreads();
    
    // Process pixel - write directly to global memory
    if (row < INPUT_HEIGHT && col < INPUT_WIDTH) {
        uint32_t value = input_tile[tid + (threadIdx.y)];
        atomicAdd(&dBins[value], 1);
    }
}

__global__ void ParallelStrategySeven(uint32_t* dInput, uint32_t* dBins)
{
    // Optimize shared memory layout - use linear indexing
    __shared__ uint32_t input_tile[1024];  // 32*32 without extra padding
    
    // Calculate indices
    int col = blockDim.x * blockIdx.x + threadIdx.x;
    int row = blockDim.y * blockIdx.y + threadIdx.y;
    int tid = threadIdx.y * blockDim.x + threadIdx.x;
    
    // Calculate global linear index for coalesced memory access
    int globalIdx = col + (row * PAD_WIDTH);
    
    // Load input tile - simplified indexing
    if (row < INPUT_HEIGHT && col < INPUT_WIDTH) {
        input_tile[tid] = dInput[globalIdx];
    }
    __syncthreads();
    
    // Process pixel - same direct atomic update
    if (row < INPUT_HEIGHT && col < INPUT_WIDTH) {
        uint32_t value = input_tile[tid];
        atomicAdd(&dBins[value], 1);
    }
}

__global__ void ParallelStrategyEight(uint32_t* dInput, uint32_t* dBins)
{
    // Original shared memory
    __shared__ uint32_t input_tile[1024];  // 32*32 without extra padding
    
    // Add local histogram bins
    __shared__ uint32_t local_histogram[256];

    // Calculate indices
    int col = blockDim.x * blockIdx.x + threadIdx.x;
    int row = blockDim.y * blockIdx.y + threadIdx.y;
    int tid = threadIdx.y * blockDim.x + threadIdx.x;
    
    // Calculate global linear index for coalesced memory access
    int globalIdx = col + (row * PAD_WIDTH);
    
    // Initialize local histogram to 0
    if(tid < 256) {
        local_histogram[tid] = 0;
    }
    __syncthreads();
    
    // Original load input tile
    if (row < INPUT_HEIGHT && col < INPUT_WIDTH) {
        input_tile[tid] = dInput[globalIdx];
    }
    __syncthreads();
    
    // Original process pixel
    if (row < INPUT_HEIGHT && col < INPUT_WIDTH) {
        uint32_t value = input_tile[tid];
        atomicAdd(&dBins[value], 1);
    }
}

__global__ void ParallelStrategyNine(uint32_t* dInput, uint32_t* dBins)
{
    // Original shared memory
    __shared__ uint32_t input_tile[1024];  // 32*32 without extra padding
    __shared__ uint32_t localBins[1024];    // One bin per value

    // Calculate indices
    int col = blockDim.x * blockIdx.x + threadIdx.x;
    int row = blockDim.y * blockIdx.y + threadIdx.y;
    int tid = threadIdx.y * blockDim.x + threadIdx.x;
    int globalIdx = col + (row * PAD_WIDTH);

    // Initialize local bins
    if(tid < 1024) {
        localBins[tid] = 0;
    }
    __syncthreads();

    // Load and process data
    if (row < INPUT_HEIGHT && col < INPUT_WIDTH) {
        uint32_t value = dInput[globalIdx];
        atomicAdd(&localBins[value], 1);
    }
    __syncthreads();

    // Reduction - have each thread handle its own bin
    if(tid < 1024) {
        uint32_t count = localBins[tid];
        if(count > 0) {
            atomicAdd(&dBins[tid], count);
        }
    }
}

int ceilDiv(int a, int b)
{
    return (a + b - 1) / b;
}

// New function that selects which strategy to run
void opt_2dhisto_strategy(uint32_t* dInput, uint32_t* dBins, int strategy)
{
    // Clear bins before running any strategy
    hipMemset(dBins, 0, HISTO_HEIGHT * HISTO_WIDTH * sizeof(uint32_t));

    // Grid and block dimensions
    dim3 gridDims(ceilDiv(PAD_WIDTH, 32), ceilDiv(INPUT_HEIGHT, 32));
    dim3 blockDims(32, 32);
    size_t sharedMemSize = 255 * sizeof(uint32_t);
    
    // Variables needed for strategy 1
    size_t num_of_cuts = 1;
    size_t num_segments_width = ceilDiv((INPUT_WIDTH), num_of_cuts);
    size_t num_segments_height = ceilDiv((INPUT_HEIGHT), num_of_cuts);
    size_t rectangle_width = (INPUT_WIDTH / num_segments_width);
    size_t rectangle_height = (INPUT_HEIGHT / num_segments_height);
    dim3 strategy1BlockDims(1, 1);
    dim3 strategy1GridDims(num_segments_width, num_segments_height);

    switch (strategy) {
        case 0: // Baseline
            Baseline<<<gridDims, blockDims>>>(dInput, dBins);
            break;
        case 1:
            // Use the variables for Strategy 1 which were already defined
            ParallelStrategyOne<<<strategy1GridDims, strategy1BlockDims>>>(dInput, dBins, rectangle_width, rectangle_height);
            break;
        case 2:
            ParallelStrategyTwo<<<gridDims, blockDims, sharedMemSize>>>(dInput, dBins);
            break;
        case 3:
            ParallelStrategyThree<<<gridDims, blockDims, sharedMemSize>>>(dInput, dBins);
            break;
        case 4:
            ParallelStrategyFour<<<gridDims, blockDims, sharedMemSize>>>(dInput, dBins);
            break;
        case 5:
            ParallelStrategyFive<<<gridDims, blockDims, sharedMemSize>>>(dInput, dBins);
            break;
        case 6:
            ParallelStrategySix<<<gridDims, blockDims, sharedMemSize>>>(dInput, dBins);
            break;
        case 7:
            ParallelStrategySeven<<<gridDims, blockDims, sharedMemSize>>>(dInput, dBins);
            break;
        case 8:
            ParallelStrategyEight<<<gridDims, blockDims, sharedMemSize>>>(dInput, dBins);
            break;
        case 9:
            ParallelStrategyNine<<<gridDims, blockDims, sharedMemSize>>>(dInput, dBins);
            break;
        default:
            // Default to strategy 9 if an invalid strategy is specified
            ParallelStrategyNine<<<gridDims, blockDims, sharedMemSize>>>(dInput, dBins);
    }

    // Ensure kernel execution is complete
    hipDeviceSynchronize();
}

// Original function now calls strategy 9 by default
void opt_2dhisto(uint32_t* dInput, uint32_t* dBins)
{
    // Just call strategy 9 (the one that was previously being used)
    opt_2dhisto_strategy(dInput, dBins, 9);
}

uint32_t* allocateAndCopyDeviceInput(uint32_t** hInput, size_t height,
                                     size_t width, size_t elementSize)
{
    size_t    dInputSize = height * PAD_WIDTH * sizeof(uint32_t);
    uint32_t* dInput;

    hipMalloc((void**) &dInput, height * PAD_WIDTH * sizeof(uint32_t));

    // correct indexing of hInput to dInput
    for (int i = 0; i < INPUT_HEIGHT; i++) {
        int offset = i * PAD_WIDTH;
        hipMemcpy(&dInput[offset], hInput[i], PAD_WIDTH * sizeof(uint32_t),
                   hipMemcpyHostToDevice);
    }

    uint32_t* dInputVerif =
            (uint32_t*) malloc(height * PAD_WIDTH * sizeof(uint32_t));
    for (int i = 0; i < INPUT_HEIGHT; i++) {
        int offset = i * PAD_WIDTH;
        hipMemcpy((void*) &dInputVerif[offset], &dInput[offset],
                   PAD_WIDTH * sizeof(uint32_t), hipMemcpyDeviceToHost);
    }
    printf("device input size is %zu\n", dInputSize);

    return dInput;
}

uint32_t* allocateAndOutputDeviceBins()
{
    uint32_t* dBins;
    hipMalloc((void**) &dBins, HISTO_HEIGHT * HISTO_WIDTH * sizeof(uint32_t));
    // first memset for initialization of bins
    hipMemset(dBins, 0, HISTO_HEIGHT * HISTO_WIDTH * sizeof(uint32_t));
    return dBins;
}

void dev_to_host_bins(uint8_t* final_bins, uint32_t* dBins)
{
    uint32_t temp_bins[HISTO_HEIGHT * HISTO_WIDTH];
    hipMemcpy(temp_bins, dBins, HISTO_HEIGHT * HISTO_WIDTH * sizeof(uint32_t),
               hipMemcpyDeviceToHost);
    // iterate through entire histogram, and limit count to 255
    for (int i = 0; i < HISTO_HEIGHT * HISTO_WIDTH; i++) {
        if (temp_bins[i] > 255)
            final_bins[i] = 255;
        else
            final_bins[i] = temp_bins[i];
    }
}

void free_device_memory(uint32_t* dInput, uint32_t* dBins)
{
    hipFree(dInput);
    hipFree(dBins);
}
